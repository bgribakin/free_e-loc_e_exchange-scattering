#include "hip/hip_runtime.h"
/*
  bla bla 
*/

#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <cstdio>
#include <cmath>
#include <ctime>
#include <cstring>
#include <conio.h>

#include <io.h>//access
#include <direct.h>//getcwd

#include "main.h"
#include "gpu_functions.h"
#include "gpuReductionSum.h"

int main() {

	time_t tic;
	time_t toc;
	time_t seed = clock() + time(0);

	constants a = {};
	constants* constants_SI, * gpu_constants_SI;
	constants_SI = &a;

	double m_e = 0.11; // eff e mass in CdTe, units of m0
	double eps = 10.2; // static dielectric constant in CdTe
	double a0 = 3e-9; // Bohr radius of the localized electron, m
	double L = 20e-9; // QW width, m
	double V_MC = init_constants_SI(constants_SI, m_e, eps, a0, L); // initializes struct pointer with constants: pi, hbar, e2eps, m_e, and a0; also calculates V_MC

	gpuErrchk(hipMalloc((void**)&gpu_constants_SI, sizeof(constants)));
	gpuErrchk(hipMemcpy(gpu_constants_SI, constants_SI, sizeof(constants), hipMemcpyHostToDevice));

	char filename[] = "integral__.dat";
	f_head_display(filename);
	head_display();

	int blockSize = 384;
	int numBlocks = (N + blockSize) / blockSize;

	hiprandState_t* states;
	gpuErrchk(hipMalloc((void**)&states, N * sizeof(hiprandState_t))); // space for random states

	tic = clock();
	initRand << <numBlocks, blockSize >> > (seed, 0, states); // invoke the GPU to initialize all of the random states
	gpuErrchk(hipDeviceSynchronize());

	double cpu_f; // variable for sum of integrand function values inside a run on cpu
	double cpu_f2; // var for the sum of it's squares (to calculate error later) on cpu

	double cpu_f_sum = 0.0; // vars to accumulate final values across all runs
	double cpu_f2_sum = 0.0;

	double temp_res; // for storing integral estimates in real-time 
	double temp_err;

	double* gpu_f; // array for integrand function values at N random points on gpu
	double* gpu_f2; // array for it's squares (to calculate error later) on gpu
	gpuErrchk(hipMalloc((void**)&gpu_f, numPoints * sizeof(double)));
	gpuErrchk(hipMalloc((void**)&gpu_f2, numPoints * sizeof(double)));

	double* gpu_f_out;
	double* gpu_f2_out;
	gpuErrchk(hipMalloc((void**)&gpu_f_out, numPoints * sizeof(double)));
	gpuErrchk(hipMalloc((void**)&gpu_f2_out, numPoints * sizeof(double)));


	double k = 0;
	double phi_k = 0;

	// main loop to accumulate integral estimate and error
	long long int runCounter;
	for (runCounter = 0; runCounter < numRun; runCounter++) {

		intMC_J_ee_exch << <numBlocks, blockSize >> > (states, gpu_constants_SI, gpu_f, gpu_f2, dim, k, phi_k); // accumulate func and func^2 evaluations in gpu_f and gpu_f2

		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		// efficient parallel reduction sum algorithm
		sumGPUDouble(gpu_f, gpu_f_out, numPoints); 
		sumGPUDouble(gpu_f2, gpu_f2_out, numPoints);

		// copy back
		gpuErrchk(hipMemcpy(&cpu_f, gpu_f, sizeof(double), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(&cpu_f2, gpu_f2, sizeof(double), hipMemcpyDeviceToHost));

		cpu_f_sum += cpu_f;
		cpu_f2_sum += cpu_f2;

		if (runCounter % 5 == 0) { //  we lose speed if we printf on every run
			toc = live_control_and_display(filename, tic, runCounter, V_MC, cpu_f_sum, cpu_f2_sum);
		}
	}

	f_data_display(filename, temp_res, temp_err, runCounter, tic, toc);

	gpuErrchk(hipFree(states));
	gpuErrchk(hipFree(gpu_f));
	gpuErrchk(hipFree(gpu_f2));
	gpuErrchk(hipFree(gpu_f_out));
	gpuErrchk(hipFree(gpu_f2_out));

	finish_display();
	
	return 1;
}